#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/ex_layers/leaky_clip_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void lClipForward(const int n, const Dtype* in, Dtype* out,
    Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
    if ( in[index] <= Dtype(-1) ) {
      out[index] = -1 + negative_slope * (in[index]+1);
    } else if ( in[index] > Dtype(-1) && in[index] < Dtype(1) ) {
      out[index] = in[index] ;
    } else {
      out[index] =  1 + negative_slope * (in[index]-1);
    }
  }
}

template <typename Dtype>
void lClipLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
  // NOLINT_NEXT_LINE(whitespace/operators)
  lClipForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, negative_slope);
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void lClipBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff, Dtype negative_slope) {
  CUDA_KERNEL_LOOP(index, n) {
      if ( in_data[index] > Dtype(-1) && in_data[index] < Dtype(1) ) {
        out_diff[index] = in_diff[index];
      } else {
        out_diff[index] = negative_slope * in_diff[index];
      }
  }
}

template <typename Dtype>
void lClipLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype negative_slope = this->layer_param_.relu_param().negative_slope();
    // NOLINT_NEXT_LINE(whitespace/operators)
    lClipBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_data, bottom_diff, negative_slope);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(lClipLayer);


}  // namespace caffe
